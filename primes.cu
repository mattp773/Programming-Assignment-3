/* From: https://github.com/mvx24 
 * Find the sum of all primes below 2 million (Project Euler #10).
 * This can take a while! *spoiler* 142913828922
 * For below 2k: 277050 (0.09s via nvcc, 19 hours via kcc!)
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 512

// Kernel that executes on the CUDA device
__global__ void sum_primes(int* firstPrimes, size_t n, unsigned long long* blockSums, int TOTAL_THREADS, int START_NUMBER) {
      __shared__ int blockPrimes[THREADS_PER_BLOCK];
      int i;
      int idx;
      int num;

      idx = blockIdx.x * blockDim.x + threadIdx.x;
      if (idx < TOTAL_THREADS) {
            // The number to test
            // printf("idx: %d\n", idx);
            if(START_NUMBER % 2 != 0 ) {
                num = START_NUMBER + (idx * 2);
            }
            else
                num = (START_NUMBER - 1) + (idx * 2);
            // printf("testing %d\n", num);
            for (i = 0; i < n; ++i) {
                  if(!(num % firstPrimes[i])) break;
            }
            if (i == n) {
                  blockPrimes[threadIdx.x] = num;
                //   printf("%d is prime\n", num);
            }
            else
                  blockPrimes[threadIdx.x] = 0;
      } else {
            blockPrimes[threadIdx.x] = 0;
      }

      __syncthreads();

      if (threadIdx.x == 0) {
            // sum all the results from the block
            blockSums[blockIdx.x] = 0;
            for (i = 0; i < blockDim.x; ++i)
                  blockSums[blockIdx.x] += blockPrimes[i];
      }
}

// main routine that executes on the host
int main(int argc, char *argv[]) {
      //host

      if(argc != 2) {
            printf("Usage: %s <number>\n", argv[0]);
            return 1;
      }

      int END_NUMBER = atoi(argv[1]);
      int START_NUMBER = (int)sqrt((double)END_NUMBER) + 1;
      const int n = pow(2, (ceil(log2(START_NUMBER)) + 1));
      const int TOTAL_THREADS = ((END_NUMBER + 2 - START_NUMBER) / 2);

    //   printf("total threads: %d\n", TOTAL_THREADS);
    //   printf("end number: %d\n", END_NUMBER);
    //     printf("start number: %d\n", START_NUMBER);
    //     printf("n: %d\n", n);

    //   if(TOTAL_THREADS < 512) {
    //     THREADS_PER_BLOCK = TOTAL_THREADS;
    //   }

      int *primes = (int *)malloc((n + 1) * sizeof(int));

      unsigned long long *primeSums;
      int i, j, index;
      int blockSize, nblocks;
      unsigned long long sum;
      size_t len;

      //device
      int* primesDevice;
      unsigned long long* primeSumsDevice;

      // Find all the primes less than the square root of 2 million ~1414
      primes[0] = 2;
      index = 1;
      sum = 2;
      for (i = 3; i != START_NUMBER; ++i) {
            for (j = 0; j != index; ++j) {
                  if (!(i % primes[j])) break;
            }
            if (j == index) {
                  primes[index++] = i;
                //   printf("%d is prime\n", i);
                  sum += i;
            }
      }
      len = index;

      hipMalloc((void**) &primesDevice, len * sizeof(int));
      hipMemcpy(primesDevice, primes, len * sizeof(int), hipMemcpyHostToDevice);

      blockSize = THREADS_PER_BLOCK;
      nblocks = TOTAL_THREADS/blockSize + !!(TOTAL_THREADS % blockSize);
    //   printf("nblocks: %d\n", nblocks);
    //   printf("blockSize: %d\n", blockSize);
      hipMalloc((void**) &primeSumsDevice, nblocks * sizeof(unsigned long long));

      sum_primes <<< nblocks, blockSize >>> (primesDevice, index, primeSumsDevice, TOTAL_THREADS, START_NUMBER);

      // Retrieve result from device and store it in host array
      primeSums = (unsigned long long*) malloc(nblocks * sizeof(unsigned long long));
      hipMemcpy(primeSums, primeSumsDevice, nblocks * sizeof(unsigned long long), hipMemcpyDeviceToHost);
      for (i = 0; i != nblocks; ++i) {
            sum += primeSums[i];
            //printf("%llu\t", primeSums[i]);
      }

      // Cleanup
      free(primeSums);
      hipFree(primeSumsDevice);
      hipFree(primesDevice);

      // Print results
      printf("Sum of primes less than %d = %llu\n",END_NUMBER, sum);
}