#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> // automatically included when compiling with nvcc

#define THREADS_PER_BLOCK 512

// kernel executed on device (GPU)
__global__ void sum_primes(int* primes, size_t n, unsigned long long* blockSums, int TOTAL_THREADS, int START_NUMBER) {
      // shared array of primes between threads of same block
      __shared__ int blockPrimes[THREADS_PER_BLOCK];
      
      // unique thread index determined by block dimensions
      int idx = blockIdx.x * blockDim.x + threadIdx.x;

      int i;
      int num;
      if (idx < TOTAL_THREADS) {
            if(START_NUMBER % 2 != 0 ) {
                num = START_NUMBER + (idx * 2);
            }
            else {
                num = (START_NUMBER - 1) + (idx * 2);
            }
            for (i = 0; i < n; ++i) {
                  if(!(num % primes[i])){ 
                        break; 
                  }
            }
            if (i == n) {
                  blockPrimes[threadIdx.x] = num; //add in prime
            }
            else {
                  blockPrimes[threadIdx.x] = 0; //not prime
            }
      } 
      else {
            blockPrimes[threadIdx.x] = 0;
      }

      __syncthreads(); // allows all threads of the block to "catch up"

      if (threadIdx.x == 0) {
            // loop and add up prime results from the block
            blockSums[blockIdx.x] = 0;
            for (i = 0; i < blockDim.x; ++i){
                  blockSums[blockIdx.x] += blockPrimes[i];
            }
      }
}

// main executed on host (CPU)
int main(int argc, char *argv[]) {
      // input error checking
      if(argc != 2) {
            printf("Usage: %s <number>\n", argv[0]);
            return 1;
      }

      // host variables
      int END_NUMBER = atoi(argv[1]);
      int START_NUMBER = (int)sqrt((double)END_NUMBER) + 1;
      const int n = pow(2, (ceil(log2(START_NUMBER)) + 1));
      const int TOTAL_THREADS = ((END_NUMBER + 2 - START_NUMBER) / 2);

      // host arrays
      int *primes = (int *)malloc((n + 1) * sizeof(int));
      unsigned long long *primeSums;

      // device arrays
      int* primesDevice;
      unsigned long long* primeSumsDevice;

      // develop primes host array to be passed to device for computation
      primes[0] = 2;
      int index = 1;
      int j;
      unsigned long long sum = 0;
      if(END_NUMBER < 3) {
            sum = 0;
      }
      else {
            sum = 2;
            for (int i = 3; i < START_NUMBER; ++i) {
                  for (j = 0; j < index; ++j) {
                        if (!(i % primes[j])) break;
                  }
                  if (j == index) {
                        primes[index++] = i;
                        sum += i;
                  }
            }
            size_t len = index;

            // initialize GPU by setting block size and number of blocks
            int blockSize = THREADS_PER_BLOCK;
            int nblocks = TOTAL_THREADS/blockSize + !!(TOTAL_THREADS % blockSize);

            // allocate prime numbers variable on device
            hipMalloc((void**) &primesDevice, len * sizeof(int));

            // copy prime numbers from host to device variable
            hipMemcpy(primesDevice, primes, len * sizeof(int), hipMemcpyHostToDevice);

            // allocate sum of prime numbers variable on device
            hipMalloc((void**) &primeSumsDevice, nblocks * sizeof(unsigned long long));

            // call the kernel with args
            sum_primes <<< nblocks, blockSize >>> (primesDevice, index, primeSumsDevice, TOTAL_THREADS, START_NUMBER);

            // allocate sum of prime numbers variable on host
            primeSums = (unsigned long long*) malloc(nblocks * sizeof(unsigned long long));
            
            // copy results from device back to host
            hipMemcpy(primeSums, primeSumsDevice, nblocks * sizeof(unsigned long long), hipMemcpyDeviceToHost);
            
            // add up the prime sums in the array of prime sums produced by the device
            for (int i = 0; i != nblocks; ++i) {
                  sum += primeSums[i];
            }

            // free allocated memory
            free(primeSums);
            hipFree(primeSumsDevice);
            hipFree(primesDevice);
      }
      // Print results
      printf("Sum of primes less than %d = %llu\n",END_NUMBER, sum);
}