
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <float.h>

typedef struct nums {
    int value;
    bool isPrime;
} nums;

#define blockSize (256)

__global__ void prime(struct nums *is_prime) {
    int id = threadIdx.x;
    if(id == 0 || id == 1) {
        is_prime[id].isPrime = false;
        is_prime[id].value = 0;
        return;
    }
    for(int i = 2; i <= (int)sqrt((float)id); i++) {
        if(id % i == 0) {
            is_prime[id].isPrime = false;
            is_prime[id].value = 0;
            return;
        }
    }
    
}

__global__ void sum_primes(struct nums *is_prime, float *sum, int N) {
    int id = threadIdx.x;
    float sum_1 = 0;
    for(int i = id; i < N; i+= blockSize) {
        // printf("value at %d is %d\n", i, is_prime[i].value);
        sum_1 += is_prime[i].value;
    }
    __shared__ float r[blockSize];
    r[id] = sum_1;
    __syncthreads();
    for(int size = blockSize/2; size > 0; size /= 2) {
        if(id < size) {
            r[id] += r[id + size];
        }
        __syncthreads();
    }
    if(id == 0) {
        *sum = r[0];
    }
}

int main(int argc, char *argv[]) {
    if(argc != 2) {
        printf("Usage: %s <number>\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    struct nums *is_prime = (nums *)malloc(N * sizeof(nums));

    for(int i = 0; i < N; i++) {
        is_prime[i].value = i;
        // printf("is_prime[%d].value = %d\n", i, is_prime[i].value);
        is_prime[i].isPrime = true;
    }

    
    float *sum = (float *)malloc(sizeof(float));
    *sum = 0;
    float *sum_gpu;
    hipMalloc((void **)&sum_gpu, sizeof(float));
    hipMemcpy(sum_gpu, sum, sizeof(float), hipMemcpyHostToDevice);

    nums *is_prime_gpu;
    hipMalloc(&is_prime_gpu, N * sizeof(nums));
    hipMemcpy(is_prime_gpu, is_prime, N * sizeof(nums), hipMemcpyHostToDevice);

    prime<<<1,N>>>(is_prime_gpu);
    hipMemcpy(is_prime, is_prime_gpu, N * sizeof(nums), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) {
       
            // printf("%d is prime\n", is_prime[i].value);
            *sum += is_prime[i].value;
    }

    // nums *is_prime_gpu_2;
    // cudaMalloc(&is_prime_gpu_2, N * sizeof(nums));
    // cudaMemcpy(is_prime_gpu_2, is_prime, N * sizeof(nums), cudaMemcpyDeviceToHost);
    // sum_primes<<<1,blockSize>>>(is_prime_gpu, sum_gpu, N);
    // cudaMemcpy(sum, sum_gpu, sizeof(float), cudaMemcpyDeviceToHost);

    printf("Sum of primes below %d: %.0f\n", N, *sum);

    // printf("Sum of primes: %f\n", sum);
}