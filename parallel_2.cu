
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <float.h>

__global__ void prime(bool *is_prime, int N) {
    int id = threadIdx.x;
    if(id < 2) {
        is_prime[id] = false;
    }
    else if(id == 2) {
        is_prime[id] = true;
    }
    else {
        for(int i = 2; i < N; i++) {
            if(id % i == 0) {
                is_prime[id] = false;
                return;
            }
        }
    }
}

__global__ void sum_primes(bool *is_prime, float *sum, int N) {
    int id = threadIdx.x;
    extern __shared__ float s[];
    if(id == 0) {
        s[id] = 0;
    }
    else if(is_prime[id]) {
        printf("%d is prime\n", id);
        s[id] = s[id - 1] + id;
    }
    else {
        s[id] = s[id - 1];
    }
    __syncthreads();
    if(id == N - 1) {
        *sum = s[id];
    }
}

int main(int argc, char *argv[]) {
    if(argc != 2) {
        printf("Usage: %s <number>\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    bool *is_prime = (bool *)malloc(N * sizeof(bool));

    for(int i = 0; i < N; i++) {
        is_prime[i] = true;
    }
    int limit = (int)sqrt(N);
    float *sum = (float *)malloc(sizeof(float));
    *sum = 0;
    float *sum_gpu;
    hipMalloc((void **)&sum_gpu, sizeof(float));
    hipMemcpy(sum_gpu, sum, sizeof(float), hipMemcpyHostToDevice);

    bool *is_prime_gpu;
    hipMalloc(&is_prime_gpu, N * sizeof(bool));
    hipMemcpy(is_prime_gpu, is_prime, N * sizeof(bool), hipMemcpyHostToDevice);

    prime<<<1,N>>>(is_prime_gpu, limit);
    hipMemcpy(is_prime, is_prime_gpu, N * sizeof(bool), hipMemcpyDeviceToHost);

    bool *is_prime_gpu_2;
    hipMalloc(&is_prime_gpu_2, N * sizeof(bool));
    hipMemcpy(is_prime_gpu_2, is_prime, N * sizeof(bool), hipMemcpyDeviceToHost);
    sum_primes<<<1,N,N>>>(is_prime_gpu, sum_gpu, N);
    hipMemcpy(sum, sum_gpu, sizeof(float), hipMemcpyDeviceToHost);

    printf("Sum of primes below %d: %f\n", N, *sum);

    // printf("Sum of primes: %f\n", sum);
}