
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


__global__ void add_set(float *arr, float *sums, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("id = %d\n", id);
    if(id < size) {
        sums[blockIdx.x] += arr[id];
        __syncthreads();
    }
}

__global__ void find_largest_sum(float *sums, float *largest_sum, int size) {
    for(int i = 0; i < size; i++) {
        if(sums[i] > *largest_sum) {
            *largest_sum = sums[i];
        }
    }
}


int main(int argc, char *argv[])
{
    if (argc != 2) {
        printf("Usage: %s <Number of elements>\n", argv[0]);
        return 1;
    }
    
    FILE* fp = fopen("input.txt", "r");
    int N = atoi(argv[1]), size = 0, i;
    float curr, *arr = (float *)malloc(sizeof(float) * size);
    
    while (!feof(fp)) {
        fscanf(fp, "%f", &curr);
        size++;
    }
    rewind(fp);

    for (i = 0; i < size; i++) {
        fscanf(fp, "%f", &arr[i]);
    }

    fclose(fp);
    printf("done reading file\n");

    int num_blocks = size - N + 1;
    int num_threads = N;

    float *sums = (float *)malloc(sizeof(float) * num_blocks);
    float *sums_gpu;
    hipMalloc(&sums_gpu, sizeof(float) * num_blocks);
    
    float *arr_gpu;
    hipMalloc(&arr_gpu, sizeof(float) * N);
    hipMemcpy(arr_gpu, arr, sizeof(float) * size, hipMemcpyHostToDevice);
    add_set<<<num_blocks, num_threads>>>(arr_gpu, sums_gpu, size);

    // cudaMemcpy(sums, sums_gpu, sizeof(float) * num_blocks, cudaMemcpyDeviceToHost);

    // float *sums_gpu_2;
    // cudaMalloc(&sums_gpu_2, sizeof(float) * num_blocks);
    // cudaMemcpy(sums_gpu_2, sums, sizeof(float) * num_blocks, cudaMemcpyDeviceToHost);

    // float *largest_sum = (float *)malloc(sizeof(float));
    // float *largest_sum_gpu;
    // cudaMalloc(&largest_sum_gpu, sizeof(float));
    // find_largest_sum<<<1,1>>>(sums_gpu_2, largest_sum_gpu, num_blocks);
    // cudaMemcpy(largest_sum, largest_sum_gpu, sizeof(float), cudaMemcpyDeviceToHost);

    // printf("Largest sum: %f\n", *largest_sum);

    return 0;
}